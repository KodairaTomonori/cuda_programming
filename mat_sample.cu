
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 2

__global__ void matMul(int A[][N], int B[][N], int *C){
    int i = threadIdx.x;
    int j = threadIdx.y;

    C[i] += A[i][j] * B[i][j];
}


int main(){

  int A[N][N] = {{1,2},{3,4}};
  int B[N][N] = {{5,6},{7,8}};
  int C[N] = {0,0};    

  int (*pA)[N], (*pB)[N], (*pC);

  hipMalloc((void**)&pA, (N*N)*sizeof(int));
  hipMalloc((void**)&pB, (N*N)*sizeof(int));
  hipMalloc((void**)&pC, (N*N)*sizeof(int));

  hipMemcpy(pA, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(pB, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(pC, C, (N)*sizeof(int), hipMemcpyHostToDevice);

  int numBlocks = 1;
  dim3 threadsPerBlock(N,N);
  matMul<<<numBlocks,threadsPerBlock>>>(pA,pB, pC);

  hipMemcpy(C, pC, (N)*sizeof(int), hipMemcpyDeviceToHost);

  int i; printf("C = \n");
  for(i=0;i<N;i++){
    printf("%d ", C[i]);
  }
  printf("\n");

  hipFree(pA); 
  hipFree(pB); 
  hipFree(pC);

  printf("\n");

  return 0;
}
